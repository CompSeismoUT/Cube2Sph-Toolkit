#include "hip/hip_runtime.h"
#include "mesh_constants_cuda.h"

__global__ static void 
kernel_update_adepml_accel(const int *CPML_to_glob,int nglob_CPML,
                          const realw *Qt,const realw* rvolume,
                          realw *__restrict__ accel)
{
    #define MXM(i,j) Qt[((iglob_CPML * NDIM) + j-1) * NDIM + i-1]
    int iglob_CPML = threadIdx.x + blockDim.x * blockIdx.x;
    if(iglob_CPML < nglob_CPML) {
        int iglob = CPML_to_glob[iglob_CPML] - 1;
        realw r = rvolume[iglob_CPML];
        realw ax = (MXM(1,1)+MXM(2,1)+MXM(3,1))  / r;
        realw ay = (MXM(1,2)+MXM(2,2)+MXM(3,2))  / r;
        realw az = (MXM(1,3)+MXM(2,3)+MXM(3,3))  / r;
        // atomicAdd(&accel[iglob*NDIM+0],ax);
        // atomicAdd(&accel[iglob*NDIM+1],ay);
        // atomicAdd(&accel[iglob*NDIM+2],az);
        accel[iglob*NDIM+0] +=  ax;
        accel[iglob*NDIM+1] +=  ay;
        accel[iglob*NDIM+2] +=  az;
    }
    #undef MXM
}

extern "C"
void include_adepml_accel_aux_device_(long *Mesh_pointer)
{
    TRACE("\tinclude_adepml_accel_aux_device");
    Mesh* mp = (Mesh*)(*Mesh_pointer);
    int nglob_CPML = mp->nglob_CPML;
    int nblocks = (nglob_CPML + BLOCKSIZE_KERNEL1 - 1) / BLOCKSIZE_KERNEL1;

    kernel_update_adepml_accel <<< nblocks,BLOCKSIZE_KERNEL1,0,mp->compute_stream >>> (
        mp->d_CPML_to_glob,nglob_CPML,mp->d_Qt,mp->d_rvolume,mp->d_accel
    );

}

__global__ static void 
kernel_update_Qu_conv(const realw *coeff_exp1, const realw* coeff_exp2,
                        int nspec_CPML, const realw* Qu_t,realw* __restrict__ Qu)
{
    int idx = blockIdx.x + gridDim.x*blockIdx.y; // = 
    int igll3 = threadIdx.x;
    if(idx < nspec_CPML * NDIM && igll3 < NGLL3){
        int ispec_CPML = idx / NDIM, idim = idx % NDIM;
        // Qu(3,3,NGLL3,nspec_CPML) coeff(3,NGLL3,nspec_CPML)
        for(int i = 0; i < NDIM; i ++) {
            int idx_q = ((ispec_CPML * NGLL3 + igll3) * NDIM + i) * NDIM + idim;
            int idx_c = (ispec_CPML * NGLL3 + igll3) * NDIM + idim;
            Qu[idx_q] = Qu[idx_q] * coeff_exp1[idx_c] + 
                        Qu_t[idx_q] * coeff_exp2[idx_c];
        }
    }
}

extern "C"
void update_qu_conv_device_(long *Mesh_pointer)
{
    TRACE("\tupdate_Qu_conv_device");
    Mesh* mp = (Mesh*)(*Mesh_pointer);
    int nspec_CPML = mp->nspec_pml;
    int nbx,nby;
    get_blocks_xy(nspec_CPML*NDIM,&nbx,&nby);
    dim3 grid(nbx,nby,1);
    dim3 block(NGLL3_PADDED,1,1);

    kernel_update_Qu_conv <<< grid,block,0,mp->compute_stream >>> (
        mp->d_coeff_exp1,mp->d_coeff_exp2,nspec_CPML,
        mp->d_Qu_t,mp->d_Qu
    );

    hipMemsetAsync(mp->d_Qu_t,0,
                    sizeof(realw)*nspec_CPML*NDIM*NDIM*NGLL3,
                mp->compute_stream);

}

/**
 * do i=1,nglob_CMPL
 * Qt_t(:,:,i) = Qt_t(:,:,i)*rvolume(i)
 */
__global__ static  void 
kernel_update_Qt_conv1(int nglob_CPML, const realw* rvolume,
                      realw* __restrict__ Qt_t)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    const int NDIM2 = NDIM * NDIM;
    if(idx < nglob_CPML * NDIM2) {
        int iglob_CPML = idx / NDIM2;
        int i = idx % NDIM2;
        int idx_q = (iglob_CPML * NDIM2 + i);
        Qt_t[idx_q] *= rvolume[iglob_CPML];
    }
}

/**
 *   do i = 1, 3 Qt(:,i,:)=Qt(:,i,:)*coeff_glob_exp1(:,:) + coeff_glob_exp2(:,:)*Qt_t(:,i,:)
 */
__global__ static  void 
kernel_update_Qt_conv2(int nglob_CPML, const realw *coeff_glob_exp1,
                        const realw* coeff_glob_exp2,realw* __restrict__ Qt,
                        const realw* Qt_t)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < nglob_CPML * NDIM) {
        int iglob_CPML = idx / NDIM, idim = idx % NDIM;
        int idx_c = iglob_CPML * NDIM + idim;
        for(int i = 0; i < NDIM; i ++) {
            int idx_q = (iglob_CPML*NDIM+i)*NDIM + idim;
            Qt[idx_q] = Qt[idx_q] * coeff_glob_exp1[idx_c] + 
                        Qt_t[idx_q] * coeff_glob_exp2[idx_c];
        }
    }
}

extern "C" void
update_qt_conv_device_(long *Mesh_pointer)
{
    TRACE("\tupdate_Qt_conv_device");
    Mesh* mp = (Mesh*)(*Mesh_pointer);
    int nglob_CPML = mp->nglob_CPML;

    // GPU resources
    int nb1 = (nglob_CPML*NDIM + BLOCKSIZE_KERNEL1 - 1) / BLOCKSIZE_KERNEL1;
    int nb2 = (nglob_CPML*NDIM*NDIM + BLOCKSIZE_KERNEL1 - 1) / BLOCKSIZE_KERNEL1;
    
    // kernel_update_Qt_conv <<< nb,BLOCKSIZE_KERNEL1,0,mp->compute_stream >>> (
    //     nglob_CPML,mp->d_coeff_glob_exp1,mp->d_coeff_glob_exp2,mp->d_rvolume,
    //     mp->d_Qt,mp->d_Qt_t
    // );
    kernel_update_Qt_conv1 <<< nb2,BLOCKSIZE_KERNEL1,0,mp->compute_stream >>> (
        nglob_CPML,mp->d_rvolume,mp->d_Qt_t
    );
    kernel_update_Qt_conv2 <<< nb1,BLOCKSIZE_KERNEL1,0,mp->compute_stream >>> (
        nglob_CPML,mp->d_coeff_glob_exp1,mp->d_coeff_glob_exp2,
        mp->d_Qt,mp->d_Qt_t
    );

    hipMemsetAsync(mp->d_Qt_t,0,sizeof(realw)*nglob_CPML*NDIM*NDIM,mp->compute_stream);
}